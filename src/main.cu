#include "hip/hip_runtime.h"
#include "external/BMP.cuh"
#include "geometry/ray.cuh"
#include "global.cuh"
#include "cudaGlobal.cuh"
#include "vector/vec3f.cuh"
#include "scene/scene.cuh"
#include "geometry/hittablelist.cuh"
#include "geometry/sphere.cuh"
//#include "renderer/Renderer.cuh"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

__device__ color ray_color(const ray &r) {
    vec3f unit_direction = unit_vector(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.0);
    return (1.0f - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
}

__global__ void rand_init(hiprandState *rand_state, int width, int height,
                          long clock_for_rand) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(212, x, 0, &rand_state[x]);
}

// __device__ int image_width = 1200;
// __device__ int image_height = 675;

__global__ void pixel_draw(uint32_t image_width, uint32_t image_height, vec3f *cam, uint8_t *image) {
    uint32_t now = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t i = now % image_width;
    uint32_t j = now / image_width;

    auto u = float(i) / (image_width - 1);
    auto v = float(j) / (image_height - 1);
    ray r(cam[0], cam[3] + u * cam[1] + v * cam[2] - cam[0]);
    color pixel_color = ray_color(r);

    uint32_t idx = now * 3;
    image[idx + 0] = static_cast<uint8_t>(255.999 * pixel_color.b());
    image[idx + 1] = static_cast<uint8_t>(255.999 * pixel_color.g());
    image[idx + 2] = static_cast<uint8_t>(255.999 * pixel_color.r());
}

int main() {

    // Image
    const float aspect_ratio = 16.0 / 9.0;
    const int image_width = 1200;
    const int image_height = static_cast<int>(image_width / aspect_ratio);

    // Camera

    float viewport_height = 2.0;
    float viewport_width = aspect_ratio * viewport_height;
    float focal_length = 1.0;

    std::vector<vec3f> CAM(4);

    vec3f origin = point3(0, 0, 0);
    vec3f horizontal = vec3f(viewport_width, 0, 0);
    vec3f vertical = vec3f(0, viewport_height, 0);
    vec3f lower_left_corner =
            origin - horizontal / 2 - vertical / 2 - vec3f(0, 0, focal_length);

    CAM[0] = origin;
    CAM[1] = horizontal;
    CAM[2] = vertical;
    CAM[3] = lower_left_corner;



    uint32_t PhotoSize = image_width * image_height * 3 * sizeof(uint8_t);
    // grid block
    uint32_t threadsPerBlock = 256;
    uint32_t blocksPerGrid =
            (image_height * image_width + threadsPerBlock - 1) / threadsPerBlock;


    // Render
    bmp::bitmap photo(image_width, image_height); // photo
    std::cout << "Photo Size : " << image_width << ' ' << image_height << "\n";

    // CUDA
    hipError_t err = hipSuccess;

    // CUDA rand

    // long clock_for_rand = clock();
    // hiprandState* rand_state;
    // err = hipMalloc((void**)&rand_state, PhotoSize * sizeof(hiprandState));
    // CudaAllocErrorMSG(err, "rand_state");
    // rand_init<<<blocksPerGrid, threadsPerBlock>>>(rand_state, image_width,
    // image_height, clock_for_rand);

    // CUAD camera
    vec3f *cameraInGPU = nullptr;
    err = hipMalloc((void **) &cameraInGPU, 4 * sizeof(vec3f));
    CudaAllocErrorMSG(err, "cameraInGPU");
    err = hipMemcpy(cameraInGPU, CAM.data(), 4 * sizeof(vec3f), hipMemcpyHostToDevice);

    // CUDA alloc
    uint8_t *photoInGPU = nullptr;
    err = hipMalloc((void **) &photoInGPU, PhotoSize);
    CudaAllocErrorMSG(err, "photoInGPU");

    pixel_draw<<<blocksPerGrid, threadsPerBlock>>>(image_width, image_height, cameraInGPU, photoInGPU);

    hipDeviceSynchronize();

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(photo.image.data(), photoInGPU, PhotoSize,
                     hipMemcpyDeviceToHost);

    CudaCopyErrorMSG(err, "Device to Host");

    hipFree(photoInGPU);

    hipDeviceSynchronize();
    hipProfilerStop();

    std::cout << (int) photo.image[10000][0] << " " << (int) photo.image[10000][1]
              << " " << (int) photo.image[10000][2] << " " << std::endl;

    photo.generate("test.bmp");
    return 0;
}
